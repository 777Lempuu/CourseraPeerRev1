#include <iostream>
#include <dirent.h>
#include <sys/stat.h>
#include "image_utils.hpp"
#include "gaussian_blur.hpp"

int main(int argc, char* argv[]) {
    if (argc != 3) {
        std::cerr << "Usage: ./main input_dir output_dir\n";
        return -1;
    }

    std::string input_dir = argv[1], output_dir = argv[2];

    DIR *dir;
    struct dirent *ent;
    if ((dir = opendir(input_dir.c_str())) != NULL) {
        while ((ent = readdir(dir)) != NULL) {
            std::string filename = ent->d_name;

            if (filename == "." || filename == "..") continue;

            std::string full_path = input_dir + "/" + filename;
            cv::Mat input = readImage(full_path);
            if (input.empty()) continue;

            cv::Mat output = applyGaussianBlurCUDA(input);

            std::string out_path = output_dir + "/" + filename;
            saveImage(out_path, output);
            std::cout << "Processed: " << filename << "\n";
        }
        closedir(dir);
    } else {
        perror("Cannot open input directory");
        return -1;
    }

    return 0;
}
