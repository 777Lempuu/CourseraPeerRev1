#include "hip/hip_runtime.h"
#include "gaussian_blur.hpp"
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>

__global__ void gaussianBlurKernel(unsigned char* input, unsigned char* output, int width, int height, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    int kernel[3][3] = {{1, 2, 1},
                        {2, 4, 2},
                        {1, 2, 1}};
    int sumKernel = 16;

    for (int c = 0; c < channels; c++) {
        int blur = 0;
        for (int ky = -1; ky <= 1; ky++) {
            for (int kx = -1; kx <= 1; kx++) {
                int nx = min(max(x + kx, 0), width - 1);
                int ny = min(max(y + ky, 0), height - 1);
                int idx = (ny * width + nx) * channels + c;
                blur += input[idx] * kernel[ky + 1][kx + 1];
            }
        }
        int out_idx = (y * width + x) * channels + c;
        output[out_idx] = blur / sumKernel;
    }
}

cv::Mat applyGaussianBlurCUDA(const cv::Mat& input) {
    int img_size = input.cols * input.rows * input.channels();
    unsigned char *d_in, *d_out;

    hipMalloc(&d_in, img_size);
    hipMalloc(&d_out, img_size);
    hipMemcpy(d_in, input.data, img_size, hipMemcpyHostToDevice);

    dim3 block(16, 16);
    dim3 grid((input.cols + 15) / 16, (input.rows + 15) / 16);
    gaussianBlurKernel<<<grid, block>>>(d_in, d_out, input.cols, input.rows, input.channels());

    cv::Mat output(input.size(), input.type());
    hipMemcpy(output.data, d_out, img_size, hipMemcpyDeviceToHost);

    hipFree(d_in); hipFree(d_out);
    return output;
}
